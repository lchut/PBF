#include "hip/hip_runtime.h"
#include "cudaPBFSolver.cuh"
#include <stdio.h>

__global__ void updateParticles(Vec3* position, Vec3* velocity, Vec3* oldPosition, int nParticles, float dt, const AABB_* boundary) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= nParticles) { return; }
    oldPosition[idx] = position[idx];
    velocity[idx] += dt * Vec3(0.0f, -49.0f, 0.0f);
    position[idx] += dt * velocity[idx];
    BoundParticle(position[idx], *boundary);

}

__global__ void updatePosition(Vec3* position, Vec3* velocity, const Vec3* oldPosition, const Vec3* offset, int nParticles, float dt, const AABB_* boundary) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= nParticles) { return; }
    position[idx] += offset[idx];
    BoundParticle(position[idx], *boundary);
    velocity[idx] = (position[idx] - oldPosition[idx]) / dt;
}

__global__ void applyVorticityAndViscosity(Vec3* velocity, const Vec3* viscosity, const Vec3* vorticity, int nParticles, float dt) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= nParticles) { return; }
    velocity[idx] += dt * vorticity[idx] + viscosity[idx];
}

__global__ void calculateParticlesGid(int* gridParticlesCnt, const Vec3* position, int nParticles, float h, const AABB_* gridSpace) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= nParticles) { return; }
    float invH = 1.0f / h;
    int gid = computeGid(position[idx], gridSpace->pMin, invH);
    if (gid >= 0) {
        atomicAdd(&gridParticlesCnt[gid], 1);
    }
}

__global__ void calculatePrefixSum(int* indata, int* outdata, int* blockSum) {
    __shared__ int temp[SHARED_MEMORY_SIZE];
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + tid;
    if (tid == 0) {
        for (int i = 0; i < SHARED_MEMORY_SIZE; ++i) {
            temp[i] = 0;
        }
    }
    __syncthreads();
    int n = blockDim.x * blockDim.y * 2;
    temp[2 * tid] = indata[2 * idx];
    temp[2 * tid + 1] = indata[2 * idx + 1];
    int offset = 1;
    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (tid < d) {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    __syncthreads();
    int blockTotalSum = 0;
    if (tid == 0) {
        blockTotalSum = temp[n - 1];
        temp[n - 1] = 0;
    }
    for (int d = 1; d < n; d *= 2) {
        __syncthreads();
        offset >>= 1;
        if (tid < d) {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    outdata[2 * idx] = temp[2 * tid];
    outdata[2 * idx + 1] = temp[2 * tid + 1];
    if (tid == 0 && blockSum != nullptr) {
        blockSum[blockId] = blockTotalSum;
    }
}

__global__ void addBlockSum(int* prefixSum, int* blockSum, int N) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= N) { return; }
    prefixSum[idx] += blockSum[blockId];
}


__global__ void setGrid(Grid_* grid, const int* gridParticlesCnt, const int* prefixSum, int N) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= N) { return; }
    if (idx == N-1) {
        grid[idx].startIdx = prefixSum[idx];
        grid[idx].endIdx = prefixSum[idx] + gridParticlesCnt[N-1];
    }
    else {
        grid[idx].startIdx = prefixSum[idx];
        grid[idx].endIdx = prefixSum[idx + 1];
    }
}
__global__ void countingSort(int* gridParticlesID, int* gridParticlesCnt, const Grid_* grid, const Vec3* position, int nParticles, float h, const AABB_* gridSpace) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= nParticles) { return; }
    float invH = 1.0f / h;
    int gid = computeGid(position[idx], gridSpace->pMin, invH);
    if (gid >= 0) {
        int offset = atomicSub(&gridParticlesCnt[gid], 1);
        gridParticlesID[grid[gid].startIdx + offset - 1] = idx;
    }
}

__global__ void buildNeighborhood(int* neighbors, const int* gridParticlesID, const Vec3* position, const Grid_* grid, int nParticles, float h, const AABB_* gridSpace) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= nParticles) { return; }
    int* pNeighbor = &neighbors[idx * (MAX_NEIGHBOR + 1)];
    int neighborCnt = 0;
    int dim[3];
    computeDimID(position[idx], gridSpace->pMin, 1.0f / h, &dim[0], &dim[1], &dim[2]);
    int neiborDim[3];
    for (int j = 0; j < 27 && neighborCnt < MAX_NEIGHBOR; ++j) {
        int xOffset = j % 3 - 1;
        int yOffSet = int(j / 3) % 3 - 1;
        int zOffset = int(j / 9) - 1;
        neiborDim[0] = dim[0] + xOffset;
        neiborDim[1] = dim[1] + yOffSet;
        neiborDim[2] = dim[2] + zOffset;
        int gid = computeGid(neiborDim);
        if (gid >=0) {
            for (int k = grid[gid].startIdx; k < grid[gid].endIdx && neighborCnt < MAX_NEIGHBOR; ++k) {
                int nid = gridParticlesID[k];
                if (nid != idx) {
                    Vec3 r = position[idx] - position[nid];
                    if (dot(r, r) <= h * h) {
                        pNeighbor[neighborCnt++] = nid;
                    }
                }
            }
        }
    }
    pNeighbor[MAX_NEIGHBOR] = neighborCnt;
}

__global__ void calculateLambda(float* lambda, const Vec3* position, const int* neighbors, int nParticles, float h, float restRho, float epsilon) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= nParticles) { return; }
    float rho = 0.0f;
    float gradMagSum = 0;
    Vec3 gradSum(0.0f);
    const int* pNeighbors = &neighbors[idx*(MAX_NEIGHBOR + 1)];
    int neighborsCnt = pNeighbors[MAX_NEIGHBOR];
    for (int j = 0; j < neighborsCnt; ++j) {
        int nid = pNeighbors[j];
        if (nid < 0 || nid >= nParticles) { printf("%d\n", nid); }
        Vec3 r = position[idx] - position[nid];

        rho += Wpoly6(r , h);
        Vec3 grad = WspikyGrad(r, h);
        gradSum += grad;
        gradMagSum += dot(grad, grad);
    }
    //printf("%f\n", rho);
    float Ci = rho / restRho - 1;
    gradMagSum += dot(gradSum, gradSum);
    gradMagSum /= restRho * restRho;
    lambda[idx] = -Ci / (gradMagSum + epsilon);
}

__global__ void calculateOffset(Vec3* offset, const Vec3* position, const float* lambda, const int* neighbors,
                                const SolverParameter para) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= para.nParticles) { return; }
    Vec3 pOffset(0.0f);
    const int* pNeighbors = &neighbors[idx*(MAX_NEIGHBOR + 1)];
    int neighborsCnt = pNeighbors[MAX_NEIGHBOR];
    for (int j = 0; j < neighborsCnt; ++j) {
        int nid = pNeighbors[j];
        Vec3 r = position[idx] - position[nid];
        float kernelVal = Wpoly6(r, para.h);
        Vec3 kernelGrad = WspikyGrad(r, para.h);
        pOffset += (lambda[idx] + lambda[nid] + Scorr(r, para.k, para.h, para.n, para.deltaq)) * kernelGrad;
    }
    pOffset /= para.restRho;
    offset[idx] = pOffset;
}

__global__ void calculateCurl(Vec3* curl, const Vec3* position, const Vec3* velocity, const int* neighbors,
                                int nParticles, float h) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= nParticles) { return; }
    Vec3 pCurl(0.0f);
    const int* pNeighbors = &neighbors[idx*(MAX_NEIGHBOR + 1)];
    int neighborsCnt = pNeighbors[MAX_NEIGHBOR];
    for (int j = 0; j < neighborsCnt; ++j) {
        int nid = pNeighbors[j];
        Vec3 vij = velocity[nid] - velocity[idx];
        pCurl += cross(vij, WspikyGrad(position[idx] - position[nid], h));
    }
    curl[idx] = pCurl;
}

__global__ void calculateVorticityAndViscosity(Vec3* vorticity ,Vec3* viscosity,
                                                const Vec3* position, const Vec3* velocity, const Vec3* curl,
                                                const int* neighbors, int nParticles, float h, float c, float vorticityConfinementEpilon) {
    unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (idx >= nParticles) { return; }
    Vec3 pViscosity(0.0f);
    Vec3 eta(0.0f);
    const int* pNeighbors = &neighbors[idx*(MAX_NEIGHBOR + 1)];
    int neighborsCnt = pNeighbors[MAX_NEIGHBOR];
    for (int j = 0; j < neighborsCnt; ++j) {
        int nid = pNeighbors[j];
        Vec3 vij = velocity[idx] - velocity[nid];
        Vec3 r = position[idx] - position[nid];
        pViscosity -= vij * Wpoly6(r, h);
        eta += length(curl[nid]) * WspikyGrad(position[idx] - position[nid], h);
    }
    if (fabsf(length(eta)) > 1e-3) {
        eta = normalize(eta);
    }
    vorticity[idx] = fVorticity(eta, curl[idx], vorticityConfinementEpilon);
    viscosity[idx] = c * pViscosity;
}
