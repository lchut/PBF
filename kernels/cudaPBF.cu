#include "cudaPBF.h"
#include "cudaPBFSolver.cuh"
#include <chrono>
void initCUDASolverDataBuffer(const std::vector<glm::vec3>& position, const std::vector<glm::vec3>& velocity,
    SolverData& data, const SolverParameter& para,
    const AABB& gridSpace, const AABB& boundary) {
    //CHECK_CUDA(hipMalloc((void**)&data.position, para.nParticles * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&data.velocity, para.nParticles * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&data.oldPosition, para.nParticles * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&data.lambda, para.nParticles * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&data.particlesOffset, para.nParticles * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&data.particlesCurl, para.nParticles * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&data.particlesVorticity, para.nParticles * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&data.particlesViscosity, para.nParticles * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&data.neighbors, para.nParticles * (MAX_NEIGHBOR + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&data.gridSpace, 6 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&data.boundary, 6 * sizeof(float)));
    unsigned int gridCnt = UNIFORM_GRID_DIM_SIZE_X * UNIFORM_GRID_DIM_SIZE_Y * UNIFORM_GRID_DIM_SIZE_Z;
    unsigned int blockCnt = gridCnt / (PREFIX_BLOCK_DIM_X * PREFIX_BLOCK_DIM_Y);
    CHECK_CUDA(hipMalloc((void**)&data.grid, gridCnt * sizeof(Grid_)));
    CHECK_CUDA(hipMalloc((void**)&data.prefixSum, gridCnt * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&data.blockSum, blockCnt * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&data.gridParticlesCnt, gridCnt * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&data.gridParticlesID, para.nParticles * sizeof(int)));

    hipMemcpy(data.position, position.data(), para.nParticles * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data.velocity, velocity.data(), para.nParticles * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data.gridSpace, &gridSpace, 6 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data.boundary, &boundary, 6 * sizeof(float), hipMemcpyHostToDevice);
}

void freeCUDASolverDataBuffer(SolverData& data) {
    CHECK_CUDA(hipFree(data.velocity));
    CHECK_CUDA(hipFree(data.oldPosition));
    CHECK_CUDA(hipFree(data.lambda));
    CHECK_CUDA(hipFree(data.particlesOffset));
    CHECK_CUDA(hipFree(data.particlesCurl));
    CHECK_CUDA(hipFree(data.particlesVorticity));
    CHECK_CUDA(hipFree(data.particlesViscosity));
    CHECK_CUDA(hipFree(data.neighbors));
    CHECK_CUDA(hipFree(data.gridSpace));
    CHECK_CUDA(hipFree(data.boundary));
    CHECK_CUDA(hipFree(data.grid));
    CHECK_CUDA(hipFree(data.prefixSum));
    CHECK_CUDA(hipFree(data.blockSum));
    CHECK_CUDA(hipFree(data.gridParticlesCnt));
    CHECK_CUDA(hipFree(data.gridParticlesID));
}

void cudaPBFSolve(SolverData& data, const SolverParameter& para) {

    unsigned int unifromGridCnt = UNIFORM_GRID_DIM_SIZE_X * UNIFORM_GRID_DIM_SIZE_Y * UNIFORM_GRID_DIM_SIZE_Z;
    hipMemset(data.gridParticlesCnt, 0, sizeof(int) * unifromGridCnt);
    int N = para.nParticles;
    dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
    int blockSize = BLOCK_DIM_X * BLOCK_DIM_Y;
    dim3 grid((N + blockSize - 1) / blockSize, 1);
    // update position & velocity
    int* temp = new int[unifromGridCnt];
    int* gpuResult = new int[unifromGridCnt];

    updateParticles<<<grid, block>>>(data.position, data.velocity, data.oldPosition, para.nParticles, para.dt, data.boundary);
    CHECK_CUDA(hipDeviceSynchronize());
    // build neiborhood

    calculateParticlesGid<<<grid, block>>>(data.gridParticlesCnt, data.position, para.nParticles, para.h, data.gridSpace);
    CHECK_CUDA(hipDeviceSynchronize());

    int prefixBlockSize = PREFIX_BLOCK_DIM_X * PREFIX_BLOCK_DIM_Y;
    int blockCnt = (unifromGridCnt + prefixBlockSize - 1) / prefixBlockSize;
    calculatePrefixSum <<< dim3(blockCnt, 1), dim3(PREFIX_BLOCK_DIM_X >> 1, PREFIX_BLOCK_DIM_Y) >>> (data.gridParticlesCnt, data.prefixSum, data.blockSum);
    CHECK_CUDA(hipDeviceSynchronize());

    calculatePrefixSum<<<dim3(1, 1), dim3(32, blockCnt / 64)>>>(data.blockSum, data.blockSum, nullptr);
    CHECK_CUDA(hipDeviceSynchronize());

    addBlockSum<<<dim3(blockCnt, 1), dim3(PREFIX_BLOCK_DIM_X, PREFIX_BLOCK_DIM_Y) >> >(data.prefixSum, data.blockSum, unifromGridCnt);
    CHECK_CUDA(hipDeviceSynchronize());

    setGrid<<<dim3(blockCnt, 1), dim3(PREFIX_BLOCK_DIM_X, PREFIX_BLOCK_DIM_Y) >>>(data.grid, data.gridParticlesCnt, data.prefixSum, unifromGridCnt);
    CHECK_CUDA(hipDeviceSynchronize());

    countingSort<<<grid, block>>>(data.gridParticlesID, data.gridParticlesCnt, data.grid, data.position, para.nParticles, para.h, data.gridSpace);
    CHECK_CUDA(hipDeviceSynchronize());

    buildNeighborhood<<<grid, block>>>(data.neighbors, data.gridParticlesID, data.position, data.grid, para.nParticles, para.h, data.gridSpace);
    CHECK_CUDA(hipDeviceSynchronize());

    int itrCnt = 0;
    while(itrCnt++ < para.iterations) {
        // calculate Lambda
        calculateLambda<<<grid, block>>>(data.lambda, data.position, data.neighbors, para.nParticles, para.h, para.restRho, para.epsilon);
        CHECK_CUDA(hipDeviceSynchronize());
        // calculate offset
        calculateOffset<<<grid, block>>>(data.particlesOffset, data.position, data.lambda, data.neighbors, para);
        CHECK_CUDA(hipDeviceSynchronize());
        // update position & collision process
        updatePosition<<<grid, block>>>(data.position, data.velocity, data.oldPosition, data.particlesOffset, para.nParticles, para.dt, data.boundary);
        CHECK_CUDA(hipDeviceSynchronize());
    }

    //
    // calculate curl
    calculateCurl<<<grid, block>>>(data.particlesCurl, data.position, data.velocity, data.neighbors, para.nParticles, para.h);
    CHECK_CUDA(hipDeviceSynchronize());
    // calculate vorticity & XSPH viscosity
    calculateVorticityAndViscosity<<<grid, block>>>(data.particlesVorticity, data.particlesViscosity,
                                data.position, data.velocity, data.particlesCurl, data.neighbors,
                                para.nParticles, para.h, para.c, para.vorticityConfinementEpilon);
    CHECK_CUDA(hipDeviceSynchronize());
    // apply vorticity confinement and XSPH viscosity
    applyVorticityAndViscosity<<<grid, block>>>(data.velocity, data.particlesViscosity, data.particlesVorticity, para.nParticles, para.dt);
    CHECK_CUDA(hipDeviceSynchronize());

}